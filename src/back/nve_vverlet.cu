#include "hip/hip_runtime.h"
/*
   File name: nve_leapfrog.cu
   Date:      2009/04/03 13:49
   Author:    Aaron Thompson and Lukas Vlcek

   Copyright (C) 2009 Aaron Thompson and Lukas Vlcek

   This program is free software; you can redistribute it and/or
   modify it under the terms of the GNU General Public License as
   published by the Free Software Foundation; either version 2 of the
   License, or (at your option) any later version.

   This program is distributed in the hope that it will be useful, but
   WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
   General Public License for more details.

   You should have received a copy of the GNU General Public License
   in a file called COPYING along with this program; if not, write to
   the Free Software Foundation, Inc., 675 Mass Ave, Cambridge, MA
   02139, USA.
*/

#ifdef __DEVICE_EMULATION__
#include <stdio.h>
#endif

#include "nve_leapfrog.h"

// Called for each particle i=0 to natom
__global__ void vverlet_nve_pre( float4 *pos, float4 *vel, float4 *force )
{
	int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	if( i < natom )
	{       // VELOCITY VERLET //
            float4 veli = vel[i];
            float4 f = force[i];
#ifdef __DEVICE_EMULATION__
        printf("Massif: %e %e %e %e %e %e %e %e\n", massif, f.x, veli.x, veli.y, veli.z, f.x*massif, f.y*massif, f.z*massif);
#endif
            float4 posi = pos[i];
            posi.x += (veli.x + f.x)*dt;
            posi.x -= box.x *truncf(posi.x*2.0f*boxi.x );
            posi.y += (veli.y + f.y)*dt;
            posi.y -= box.y *truncf(posi.y*2.0f*boxi.y);
            posi.z += (veli.z + f.z)*dt;
            posi.z -= box.z *truncf(posi.z*2.0f*boxi.z );
            pos[i] = posi;

            veli.x += f.x;
            veli.y += f.y;
            veli.z += f.z;
            vel[i] = veli;
	}
}

// Called for each particle i=0 to natom
__global__ void vverlet_nve_cor(float4 *vel, float4 *force )
{
	int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	if( i < natom )
	{       // VELOCITY VERLET //
            float4 veli = vel[i];
            float massif = __fdividef(0.5*dt, veli.w);
            float4 f = force[i];
#ifdef __DEVICE_EMULATION__
        printf("Massif: %e %e %e %e %e %e %e %e\n", massif, f.x, veli.x, veli.y, veli.z, f.x*massif, f.y*massif, f.z*massif);
#endif
            f.x *= massif; 
            f.y *= massif;
            f.z *= massif;

            veli.x += f.x;
            veli.y += f.y;
            veli.z += f.z;
            vel[i] = veli;
	}
}


/* end of nve_leapfrog.cu */
